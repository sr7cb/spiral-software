/*
 *  Copyright (c) 2018-2021, Carnegie Mellon University
 *  See LICENSE for details
 */
/***************************************************************************
 * SPL Matrix                                                              *
 *                                                                         *
 * Computes matrix that corresponds to SPL generated routine               *
 ***************************************************************************/

#include <limits.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include <hip/hip_runtime_api.h>

#ifndef ROWS
#error ROWS must be defined
#endif
#ifndef COLUMNS
#error COLUMNS must be defined
#endif

hipfftDoubleReal  *Input, *Output;
hipfftDoubleReal  *dev_in, *dev_out;

void initialize(int argc, char **argv) {
	hipHostMalloc ( &Input,  sizeof(hipfftDoubleReal) * COLUMNS );
	hipHostMalloc ( &Output, sizeof(hipfftDoubleReal) * ROWS );

	hipMalloc     ( &dev_in,  sizeof(hipfftDoubleReal) * COLUMNS );
	hipMalloc     ( &dev_out, sizeof(hipfftDoubleReal) * ROWS );

	INITFUNC();
}

void finalize() {
	hipHostFree (Output);
	hipHostFree (Input);
	hipFree     (dev_out);
	hipFree     (dev_in);
}

void set_value_in_vector(hipfftDoubleReal *arr, int elem)
{
	// Zero array and put '1' in the location indicated by element
	int idx;
	for (idx = 0; idx < COLUMNS; idx++)
		arr[idx] = (idx == elem) ? 1.0 : 0.0;

	return;
}

void compute_matrix()
{
	int x, y;
	printf("[ ");
	for (x = 0; x < COLUMNS; x++) {
		set_value_in_vector(Input, x);

		hipMemcpy ( dev_in, Input, sizeof(hipfftDoubleReal) * COLUMNS, hipMemcpyHostToDevice);
		FUNC(dev_out, dev_in);
		hipMemcpy ( Output, dev_out, sizeof(hipfftDoubleReal) * ROWS, hipMemcpyDeviceToHost);
		
		if (x != 0) {
			printf(",\n  [ ");
		}
		else {
			printf("[ ");
		}
		for (y = 0; y < ROWS; y++) {
			if (y != 0) {
				printf(", ");
			}
			printf("FloatString(\"%.18g\")", Output[y]);
		}
		printf(" ]");
	}
	printf("\n];\n");
}


int main(int argc, char** argv) {
	initialize(argc, argv);
	compute_matrix();
	finalize();
	return EXIT_SUCCESS;
}
